
#include <hip/hip_runtime.h>

#define MIN(a,b) (((a)<=(b))? (a):(b))
#define TILE_SIZE 16

__global__ void ab_gpu_i4j4db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int j = blockIdx.x*blockDim.x*4 + tx;
	const unsigned int i = blockIdx.y*blockDim.y*4 + ty;

	float sum0000 = 0.0;
	float sum0001 = 0.0;
	float sum0010 = 0.0;
	float sum0011 = 0.0;
	float sum0100 = 0.0;
	float sum0101 = 0.0;
	float sum0110 = 0.0;
	float sum0111 = 0.0;
	float sum1000 = 0.0;
	float sum1001 = 0.0;
	float sum1010 = 0.0;
	float sum1011 = 0.0;
	float sum1100 = 0.0;
	float sum1101 = 0.0;
	float sum1110 = 0.0;
	float sum1111 = 0.0;
	__shared__ float as00[2][TILE_SIZE][TILE_SIZE];
	__shared__ float as01[2][TILE_SIZE][TILE_SIZE];
	__shared__ float as10[2][TILE_SIZE][TILE_SIZE];
	__shared__ float as11[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs00[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs01[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs10[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs11[2][TILE_SIZE][TILE_SIZE];
	unsigned short int current = 0;

	if (i<Ni && tx<Nk)	as00[0][ty][tx] = A[i*Nk + tx];
	else	as00[0][ty][tx] = 0;

	if (i+TILE_SIZE<Ni && tx<Nk)	as01[0][ty][tx] = A[(i+TILE_SIZE)*Nk + tx];
	else	as01[0][ty][tx] = 0;

	if (i+2*TILE_SIZE<Ni && tx<Nk)	as10[0][ty][tx] = A[(i+2*TILE_SIZE)*Nk + tx];
	else	as10[0][ty][tx] = 0;

	if (i+3*TILE_SIZE<Ni && tx<Nk)	as11[0][ty][tx] = A[(i+3*TILE_SIZE)*Nk + tx];
	else	as11[0][ty][tx] = 0;

	if (j<Nj && ty<Nk)	bs00[0][ty][tx] = B[ty*Nj + j];
	else	bs00[0][ty][tx] = 0;

	if (j+TILE_SIZE<Nj && ty<Nk)	bs01[0][ty][tx] = B[ty*Nj + j+TILE_SIZE];
	else	bs01[0][ty][tx] = 0;

	if (j+2*TILE_SIZE<Nj && ty<Nk)	bs10[0][ty][tx] = B[ty*Nj + j+2*TILE_SIZE];
	else	bs10[0][ty][tx] = 0;

	if (j+3*TILE_SIZE<Nj && ty<Nk)	bs11[0][ty][tx] = B[ty*Nj + j+3*TILE_SIZE];
	else	bs11[0][ty][tx] = 0;
	__syncthreads();

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0000 += as00[current][ty][k-kt]*bs00[current][k-kt][tx];
	
		if (i<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0001 += as00[current][ty][k-kt]*bs01[current][k-kt][tx];

		if (i<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0010 += as00[current][ty][k-kt]*bs10[current][k-kt][tx];

		if (i<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0011 += as00[current][ty][k-kt]*bs11[current][k-kt][tx];

		if (i+TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0100 += as01[current][ty][k-kt]*bs00[current][k-kt][tx];

		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0101 += as01[current][ty][k-kt]*bs01[current][k-kt][tx];

		if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0110 += as01[current][ty][k-kt]*bs10[current][k-kt][tx];

		if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0111 += as01[current][ty][k-kt]*bs11[current][k-kt][tx];

		if (i+2*TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1000 += as10[current][ty][k-kt]*bs00[current][k-kt][tx];

		if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1001 += as10[current][ty][k-kt]*bs01[current][k-kt][tx];
		
		if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj) 
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1010 += as10[current][ty][k-kt]*bs10[current][k-kt][tx];
		
		if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1011 += as10[current][ty][k-kt]*bs11[current][k-kt][tx];
		
		if (i+3*TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1100 += as11[current][ty][k-kt]*bs00[current][k-kt][tx];
		
		if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1101 += as11[current][ty][k-kt]*bs01[current][k-kt][tx];
		
		if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1110 += as11[current][ty][k-kt]*bs10[current][k-kt][tx];
		
		if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1111 += as11[current][ty][k-kt]*bs11[current][k-kt][tx];

		current = (current+1)&1;
		if (i<Ni && kt+TILE_SIZE+tx<Nk)
			as00[current][ty][tx] = A[i*Nk + kt+TILE_SIZE+tx];
		else
			as00[current][ty][tx] = 0;

		if (i+TILE_SIZE<Ni && kt+TILE_SIZE+tx<Nk)
			as01[current][ty][tx] = A[(i+TILE_SIZE)*Nk + kt+TILE_SIZE+tx];
		else
			as01[current][ty][tx] = 0;

		if (i+2*TILE_SIZE<Ni && kt+TILE_SIZE+tx<Nk)
			as10[current][ty][tx] = A[(i+2*TILE_SIZE)*Nk + kt+TILE_SIZE+tx];
		else
			as10[current][ty][tx] = 0;

		if (i+3*TILE_SIZE<Ni && kt+TILE_SIZE+tx<Nk)
			as11[current][ty][tx] = A[(i+3*TILE_SIZE)*Nk + kt+TILE_SIZE+tx];
		else
			as11[current][ty][tx] = 0;

		if (j<Nj && kt+TILE_SIZE+ty<Nk)
			bs00[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j];
		else
			bs00[current][ty][tx] = 0;

		if (j+TILE_SIZE<Nj && kt+TILE_SIZE+ty<Nk)
			bs01[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j+TILE_SIZE];
		else
			bs01[current][ty][tx] = 0;

		if (j+2*TILE_SIZE<Nj && kt+TILE_SIZE+ty<Nk)
			bs10[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j+2*TILE_SIZE];
		else
			bs10[current][ty][tx] = 0;

		if (j+3*TILE_SIZE<Nj && kt+TILE_SIZE+ty<Nk)
			bs11[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j+3*TILE_SIZE];
		else
			bs11[current][ty][tx] = 0;

		__syncthreads();
	}
	if (i<Ni && j<Nj)	C[i*Nj + j] = sum0000;
	if (i<Ni && j+TILE_SIZE<Nj)	C[i*Nj + j+TILE_SIZE] = sum0001;
	if (i<Ni && j+2*TILE_SIZE<Nj)	C[i*Nj + j+2*TILE_SIZE] = sum0010;
	if (i<Ni && j+3*TILE_SIZE<Nj)	C[i*Nj + j+3*TILE_SIZE] = sum0011;
	if (i+TILE_SIZE<Ni && j<Nj)	C[(i+TILE_SIZE)*Nj + j] = sum0100;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum0101;
	if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum0110;
	if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum0111;
	if (i+2*TILE_SIZE<Ni && j<Nj)	C[(i+2*TILE_SIZE)*Nj + j] = sum1000;
	if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1001;
	if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1010;
	if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1011;
	if (i+3*TILE_SIZE<Ni && j<Nj)	C[(i+3*TILE_SIZE)*Nj + j] = sum1100;
	if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1101;
	if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1110;
	if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1111;
}

__global__ void ab_gpu_i4j4(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int j = blockIdx.x*blockDim.x*4 + tx;
	const unsigned int i = blockIdx.y*blockDim.y*4 + ty;

	float sum0000 = 0.0;
	float sum0001 = 0.0;
	float sum0010 = 0.0;
	float sum0011 = 0.0;
	float sum0100 = 0.0;
	float sum0101 = 0.0;
	float sum0110 = 0.0;
	float sum0111 = 0.0;
	float sum1000 = 0.0;
	float sum1001 = 0.0;
	float sum1010 = 0.0;
	float sum1011 = 0.0;
	float sum1100 = 0.0;
	float sum1101 = 0.0;
	float sum1110 = 0.0;
	float sum1111 = 0.0;
	__shared__ float as00[TILE_SIZE][TILE_SIZE];
	__shared__ float as01[TILE_SIZE][TILE_SIZE];
	__shared__ float as10[TILE_SIZE][TILE_SIZE];
	__shared__ float as11[TILE_SIZE][TILE_SIZE];
	__shared__ float bs00[TILE_SIZE][TILE_SIZE];
	__shared__ float bs01[TILE_SIZE][TILE_SIZE];
	__shared__ float bs10[TILE_SIZE][TILE_SIZE];
	__shared__ float bs11[TILE_SIZE][TILE_SIZE];

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && kt+tx<Nk)
			as00[ty][tx] = A[i*Nk + kt+tx];
		else
			as00[ty][tx] = 0;

		if (i+TILE_SIZE<Ni && kt+tx<Nk)
			as01[ty][tx] = A[(i+TILE_SIZE)*Nk + kt+tx];
		else
			as01[ty][tx] = 0;

		if (i+2*TILE_SIZE<Ni && kt+tx<Nk)
			as10[ty][tx] = A[(i+2*TILE_SIZE)*Nk + kt+tx];
		else
			as10[ty][tx] = 0;

		if (i+3*TILE_SIZE<Ni && kt+tx<Nk)
			as11[ty][tx] = A[(i+3*TILE_SIZE)*Nk + kt+tx];
		else
			as11[ty][tx] = 0;

		if (j<Nj && kt+ty<Nk)
			bs00[ty][tx] = B[(kt+ty)*Nj + j];
		else
			bs00[ty][tx] = 0;

		if (j+TILE_SIZE<Nj && kt+ty<Nk)
			bs01[ty][tx] = B[(kt+ty)*Nj + j+TILE_SIZE];
		else
			bs01[ty][tx] = 0;

		if (j+2*TILE_SIZE<Nj && kt+ty<Nk)
			bs10[ty][tx] = B[(kt+ty)*Nj + j+2*TILE_SIZE];
		else
			bs10[ty][tx] = 0;

		if (j+3*TILE_SIZE<Nj && kt+ty<Nk)
			bs11[ty][tx] = B[(kt+ty)*Nj + j+3*TILE_SIZE];
		else
			bs11[ty][tx] = 0;

		__syncthreads();

		if (i<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0000 += as00[ty][k-kt]*bs00[k-kt][tx];

		if (i<Ni && j+TILE_SIZE<Nj) 
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0001 += as00[ty][k-kt]*bs01[k-kt][tx];
		
		if (i<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0010 += as00[ty][k-kt]*bs10[k-kt][tx];
		
		if (i<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0011 += as00[ty][k-kt]*bs11[k-kt][tx];
		
		if (i+TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0100 += as01[ty][k-kt]*bs00[k-kt][tx];
		
		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0101 += as01[ty][k-kt]*bs01[k-kt][tx];
		
		if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0110 += as01[ty][k-kt]*bs10[k-kt][tx];
		
		if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0111 += as01[ty][k-kt]*bs11[k-kt][tx];
		
		if (i+2*TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1000 += as10[ty][k-kt]*bs00[k-kt][tx];
		
		if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1001 += as10[ty][k-kt]*bs01[k-kt][tx];
		
		if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1010 += as10[ty][k-kt]*bs10[k-kt][tx];
		
		if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1011 += as10[ty][k-kt]*bs11[k-kt][tx];
		
		if (i+3*TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1100 += as11[ty][k-kt]*bs00[k-kt][tx];
		
		if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1101 += as11[ty][k-kt]*bs01[k-kt][tx];
		
		if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1110 += as11[ty][k-kt]*bs10[k-kt][tx];
		
		if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1111 += as11[ty][k-kt]*bs11[k-kt][tx];

		__syncthreads();
	}

	if (i<Ni && j<Nj)	C[i*Nj + j] = sum0000;
	if (i<Ni && j+TILE_SIZE<Nj)	C[i*Nj + j+TILE_SIZE] = sum0001;
	if (i<Ni && j+2*TILE_SIZE<Nj)	C[i*Nj + j+2*TILE_SIZE] = sum0010;
	if (i<Ni && j+3*TILE_SIZE<Nj)	C[i*Nj + j+3*TILE_SIZE] = sum0011;
	if (i+TILE_SIZE<Ni && j<Nj)	C[(i+TILE_SIZE)*Nj + j] = sum0100;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum0101;
	if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum0110;
	if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum0111;
	if (i+2*TILE_SIZE<Ni && j<Nj)	C[(i+2*TILE_SIZE)*Nj + j] = sum1000;
	if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1001;
	if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1010;
	if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1011;
	if (i+3*TILE_SIZE<Ni && j<Nj)	C[(i+3*TILE_SIZE)*Nj + j] = sum1100;
	if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1101;
	if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1110;
	if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1111;
}

__global__ void ab_gpu_i2j2db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int j = blockIdx.x*blockDim.x*2 + tx;
	const unsigned int i = blockIdx.y*blockDim.y*2 + ty;

	float sum00 = 0.0;
	float sum01 = 0.0;
	float sum10 = 0.0;
	float sum11 = 0.0;
	__shared__ float as0[2][TILE_SIZE][TILE_SIZE];
	__shared__ float as1[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs0[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs1[2][TILE_SIZE][TILE_SIZE];
	unsigned int current = 0;

	if (i<Ni && tx<Nk)
		as0[0][ty][tx] = A[i*Nk + tx];
	else
		as0[0][ty][tx] = 0;
	if (i+TILE_SIZE<Ni && tx<Nk)
		as1[0][ty][tx] = A[(i+TILE_SIZE)*Nk + tx];
	else
		as1[0][ty][tx] = 0;
	if (j<Nj && ty<Nk)
		bs0[0][ty][tx] = B[ty*Nj + j];
	else
		bs0[0][ty][tx] = 0;
	if (j+TILE_SIZE<Nj && ty<Nk)
		bs1[0][ty][tx] = B[ty*Nj + j+TILE_SIZE];
	else
		bs1[0][ty][tx] = 0;
	__syncthreads();

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		const unsigned int previous = current;
		current = (current+1)&1;
		if (i<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum00 += as0[previous][ty][k-kt]*bs0[previous][k-kt][tx];
		}

		if (i<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum01 += as0[previous][ty][k-kt]*bs1[previous][k-kt][tx];
		}

		if (i+TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum10 += as1[previous][ty][k-kt]*bs0[previous][k-kt][tx];
		}

		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum11 += as1[previous][ty][k-kt]*bs1[previous][k-kt][tx];
		}
		if (i<Ni && kt+TILE_SIZE+tx<Nk)
			as0[current][ty][tx] = A[i*Nk + kt+TILE_SIZE+tx];
		else
			as0[current][ty][tx] = 0;
		if (i+TILE_SIZE<Ni && kt+TILE_SIZE+tx<Nk)
			as1[current][ty][tx] = A[(i+TILE_SIZE)*Nk + kt+TILE_SIZE+tx];
		else
			as1[current][ty][tx] = 0;
		if (j<Nj && kt+TILE_SIZE+ty<Nk)
			bs0[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j];
		else
			bs0[current][ty][tx] = 0;
		if (j+TILE_SIZE<Nj && kt+TILE_SIZE+ty<Nk)
			bs1[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j+TILE_SIZE];
		else
			bs1[current][ty][tx] = 0;
		__syncthreads();
	}

	if (i<Ni && j<Nj)	C[i*Nj + j] = sum00;
	if (i<Ni && j+TILE_SIZE<Nj)	C[i*Nj + j+TILE_SIZE] = sum01;
	if (i+TILE_SIZE<Ni && j<Nj)	C[(i+TILE_SIZE)*Nj + j] = sum10;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum11;
}

__global__ void ab_gpu_db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int j = blockIdx.x*blockDim.x + tx;
	const unsigned int i = blockIdx.y*blockDim.y + ty;
	float sum = 0.0;
	unsigned int current = 0;

	__shared__ float as[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs[2][TILE_SIZE][TILE_SIZE];
	if (i<Ni && tx<Nk)	as[0][ty][tx] = A[i*Nk + tx];
	else				as[0][ty][tx] = 0;
	if (j<Nj && ty<Nk)	bs[0][ty][tx] = B[ty*Nj + j];
	else				bs[0][ty][tx] = 0;
	__syncthreads();

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum += as[current][ty][k-kt]*bs[current][k-kt][tx];

		current = (current+1)&1;
		if (i<Ni && kt+TILE_SIZE+tx<Nk)
			as[current][ty][tx] = A[i*Nk + kt+TILE_SIZE+tx];
		else
			as[current][ty][tx] = 0;
		if (j<Nj && kt+TILE_SIZE+ty<Nk)
			bs[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j];
		else
			bs[current][ty][tx] = 0;
		__syncthreads();
	}
	if (i<Ni && j<Nj)	C[i*Nj + j] = sum;
}

__global__ void ab_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int j = blockIdx.x*blockDim.x + tx;
	const unsigned int i = blockIdx.y*blockDim.y + ty;
	float sum = 0.0;
	unsigned int current = 0;

	__shared__ float as[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs[2][TILE_SIZE][TILE_SIZE];
	if (i<Ni && tx<Nk)	as[0][ty][tx] = A[i*Nk + tx];
	else	as[0][ty][tx] = 0;
	if (j<Nj && ty<Nk)	bs[0][ty][tx] = B[ty*Nj + j];
	else	bs[0][ty][tx] = 0;
	__syncthreads();

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum += as[current][ty][k-kt]*bs[current][k-kt][tx];
		}
		current = (current+1)&1;
		if (i<Ni && kt+TILE_SIZE+tx<Nk)
			as[current][ty][tx] = A[i*Nk + kt+TILE_SIZE+tx];
		else
			as[current][ty][tx] = 0;
		if (j<Nj && kt+TILE_SIZE+ty<Nk)
			bs[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j];
		else
			bs[current][ty][tx] = 0;
		__syncthreads();
	}
	if (i<Ni && j<Nj)	C[i*Nj + j] = sum;
}

__global__ void aTb_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int i = blockIdx.y*blockDim.y*4 + ty;
	const unsigned int j = blockIdx.x*blockDim.x*4 + tx;

	__shared__ float as00[TILE_SIZE][TILE_SIZE];
	__shared__ float as01[TILE_SIZE][TILE_SIZE];
	__shared__ float as10[TILE_SIZE][TILE_SIZE];
	__shared__ float as11[TILE_SIZE][TILE_SIZE];
	__shared__ float bs00[TILE_SIZE][TILE_SIZE];
	__shared__ float bs01[TILE_SIZE][TILE_SIZE];
	__shared__ float bs10[TILE_SIZE][TILE_SIZE];
	__shared__ float bs11[TILE_SIZE][TILE_SIZE];
	float sum0000 = 0.0;
	float sum0001 = 0.0;
	float sum0010 = 0.0;
	float sum0011 = 0.0;
	float sum0100 = 0.0;
	float sum0101 = 0.0;
	float sum0110 = 0.0;
	float sum0111 = 0.0;
	float sum1000 = 0.0;
	float sum1001 = 0.0;
	float sum1010 = 0.0;
	float sum1011 = 0.0;
	float sum1100 = 0.0;
	float sum1101 = 0.0;
	float sum1110 = 0.0;
	float sum1111 = 0.0;

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && (kt+tx)<Nk)
			as00[tx][ty] = A[(kt+tx)*Ni + i];
		else
			as00[tx][ty] = 0.0;
		if (i+TILE_SIZE<Ni && (kt+tx)<Nk)
			as01[tx][ty] = A[(kt+tx)*Ni + i+TILE_SIZE];
		else
			as01[tx][ty] = 0.0;
		if (i+2*TILE_SIZE<Ni && (kt+tx)<Nk)
			as10[tx][ty] = A[(kt+tx)*Ni + i+2*TILE_SIZE];
		else
			as10[tx][ty] = 0.0;
		if (i+3*TILE_SIZE<Ni && (kt+tx)<Nk)
			as11[tx][ty] = A[(kt+tx)*Ni + i+3*TILE_SIZE];
		else
			as11[tx][ty] = 0.0;

		if (j<Nj && (kt+ty)<Nk)
			bs00[ty][tx] = B[(kt+ty)*Nj + j];
		else
			bs00[ty][tx] = 0.0;
		if (j+TILE_SIZE<Nj && (kt+ty)<Nk)
			bs01[ty][tx] = B[(kt+ty)*Nj + j+TILE_SIZE];
		else
			bs01[ty][tx] = 0.0;
		if (j+2*TILE_SIZE<Nj && (kt+ty)<Nk)
			bs10[ty][tx] = B[(kt+ty)*Nj + j+2*TILE_SIZE];
		else
			bs10[ty][tx] = 0.0;
		if (j+3*TILE_SIZE<Nj && (kt+ty)<Nk)
			bs11[ty][tx] = B[(kt+ty)*Nj + j+3*TILE_SIZE];
		else
			bs11[ty][tx] = 0.0;
		__syncthreads();

		if (i<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0000 += as00[k-kt][ty]*bs00[k-kt][tx];
		}
		if (i<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0001 += as00[k-kt][ty]*bs01[k-kt][tx];
		}
		if (i<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0010 += as00[k-kt][ty]*bs10[k-kt][tx];
		}
		if (i<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0011 += as00[k-kt][ty]*bs11[k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0100 += as01[k-kt][ty]*bs00[k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0101 += as01[k-kt][ty]*bs01[k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0110 += as01[k-kt][ty]*bs10[k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0111 += as01[k-kt][ty]*bs11[k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1000 += as10[k-kt][ty]*bs00[k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1001 += as10[k-kt][ty]*bs01[k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1010 += as10[k-kt][ty]*bs10[k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1011 += as10[k-kt][ty]*bs11[k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1100 += as11[k-kt][ty]*bs00[k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1101 += as11[k-kt][ty]*bs01[k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1110 += as11[k-kt][ty]*bs10[k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1111 += as11[k-kt][ty]*bs11[k-kt][tx];
		}
		__syncthreads();
	}

	if (i<Ni && j<Nj)	C[i*Nj + j] = sum0000;
	if (i<Ni && j+TILE_SIZE<Nj)	C[i*Nj + j+TILE_SIZE] = sum0001;
	if (i<Ni && j+2*TILE_SIZE<Nj)	C[i*Nj + j+2*TILE_SIZE] = sum0010;
	if (i<Ni && j+3*TILE_SIZE<Nj)	C[i*Nj + j+3*TILE_SIZE] = sum0011;
	if (i+TILE_SIZE<Ni && j<Nj)	C[(i+TILE_SIZE)*Nj + j] = sum0100;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum0101;
	if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum0110;
	if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum0111;
	if (i+2*TILE_SIZE<Ni && j<Nj)	C[(i+2*TILE_SIZE)*Nj + j] = sum1000;
	if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1001;
	if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1010;
	if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1011;
	if (i+3*TILE_SIZE<Ni && j<Nj)	C[(i+3*TILE_SIZE)*Nj + j] = sum1100;
	if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1101;
	if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1110;
	if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1111;
}

__global__ void aTb_gpu_i4j4(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int i = blockIdx.y*blockDim.y*4 + ty;
	const unsigned int j = blockIdx.x*blockDim.x*4 + tx;

	__shared__ float as00[TILE_SIZE][TILE_SIZE];
	__shared__ float as01[TILE_SIZE][TILE_SIZE];
	__shared__ float as10[TILE_SIZE][TILE_SIZE];
	__shared__ float as11[TILE_SIZE][TILE_SIZE];
	__shared__ float bs00[TILE_SIZE][TILE_SIZE];
	__shared__ float bs01[TILE_SIZE][TILE_SIZE];
	__shared__ float bs10[TILE_SIZE][TILE_SIZE];
	__shared__ float bs11[TILE_SIZE][TILE_SIZE];
	float sum0000 = 0.0;
	float sum0001 = 0.0;
	float sum0010 = 0.0;
	float sum0011 = 0.0;
	float sum0100 = 0.0;
	float sum0101 = 0.0;
	float sum0110 = 0.0;
	float sum0111 = 0.0;
	float sum1000 = 0.0;
	float sum1001 = 0.0;
	float sum1010 = 0.0;
	float sum1011 = 0.0;
	float sum1100 = 0.0;
	float sum1101 = 0.0;
	float sum1110 = 0.0;
	float sum1111 = 0.0;

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && (kt+tx)<Nk)	as00[tx][ty] = A[(kt+tx)*Ni + i];
		else	as00[tx][ty] = 0.0;
		if (i+TILE_SIZE<Ni && (kt+tx)<Nk)	as01[tx][ty] = A[(kt+tx)*Ni + i+TILE_SIZE];
		else	as01[tx][ty] = 0.0;
		if (i+2*TILE_SIZE<Ni && (kt+tx)<Nk)	as10[tx][ty] = A[(kt+tx)*Ni + i+2*TILE_SIZE];
		else	as10[tx][ty] = 0.0;
		if (i+3*TILE_SIZE<Ni && (kt+tx)<Nk)	as11[tx][ty] = A[(kt+tx)*Ni + i+3*TILE_SIZE];
		else	as11[tx][ty] = 0.0;

		if (j<Nj && (kt+ty)<Nk)	bs00[ty][tx] = B[(kt+ty)*Nj + j];
		else	bs00[ty][tx] = 0.0;
		if (j+TILE_SIZE<Nj && (kt+ty)<Nk)	bs01[ty][tx] = B[(kt+ty)*Nj + j+TILE_SIZE];
		else	bs01[ty][tx] = 0.0;
		if (j+2*TILE_SIZE<Nj && (kt+ty)<Nk)	bs10[ty][tx] = B[(kt+ty)*Nj + j+2*TILE_SIZE];
		else	bs10[ty][tx] = 0.0;
		if (j+3*TILE_SIZE<Nj && (kt+ty)<Nk)	bs11[ty][tx] = B[(kt+ty)*Nj + j+3*TILE_SIZE];
		else	bs11[ty][tx] = 0.0;
		__syncthreads();

		if (i<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0000 += as00[k-kt][ty]*bs00[k-kt][tx];
		}
		if (i<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0001 += as00[k-kt][ty]*bs01[k-kt][tx];
		}
		if (i<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0010 += as00[k-kt][ty]*bs10[k-kt][tx];
		}
		if (i<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0011 += as00[k-kt][ty]*bs11[k-kt][tx];
		}

		if (i+TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0100 += as01[k-kt][ty]*bs00[k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0101 += as01[k-kt][ty]*bs01[k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0110 += as01[k-kt][ty]*bs10[k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0111 += as01[k-kt][ty]*bs11[k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1000 += as10[k-kt][ty]*bs00[k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1001 += as10[k-kt][ty]*bs01[k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1010 += as10[k-kt][ty]*bs10[k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1011 += as10[k-kt][ty]*bs11[k-kt][tx];
		}

		if (i+3*TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1100 += as11[k-kt][ty]*bs00[k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1101 += as11[k-kt][ty]*bs01[k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1110 += as11[k-kt][ty]*bs10[k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1111 += as11[k-kt][ty]*bs11[k-kt][tx];
		}
		__syncthreads();
	}
	if (i<Ni && j<Nj)	C[i*Nj + j] = sum0000;
	if (i<Ni && j+TILE_SIZE<Nj)	C[i*Nj + j+TILE_SIZE] = sum0001;
	if (i<Ni && j+2*TILE_SIZE<Nj)	C[i*Nj + j+2*TILE_SIZE] = sum0010;
	if (i<Ni && j+3*TILE_SIZE<Nj)	C[i*Nj + j+3*TILE_SIZE] = sum0011;
	if (i+TILE_SIZE<Ni && j<Nj)	C[(i+TILE_SIZE)*Nj + j] = sum0100;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum0101;
	if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum0110;
	if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum0111;
	if (i+2*TILE_SIZE<Ni && j<Nj)	C[(i+2*TILE_SIZE)*Nj + j] = sum1000;
	if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1001;
	if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1010;
	if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1011;
	if (i+3*TILE_SIZE<Ni && j<Nj)	C[(i+3*TILE_SIZE)*Nj + j] = sum1100;
	if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1101;
	if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1110;
	if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1111;
}

__global__ void aTb_gpu_i4j4db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int i = blockIdx.y*blockDim.y*4 + ty;
	const unsigned int j = blockIdx.x*blockDim.x*4 + tx;

	__shared__ float as00[2][TILE_SIZE][TILE_SIZE];
	__shared__ float as01[2][TILE_SIZE][TILE_SIZE];
	__shared__ float as10[2][TILE_SIZE][TILE_SIZE];
	__shared__ float as11[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs00[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs01[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs10[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs11[2][TILE_SIZE][TILE_SIZE];
	float sum0000 = 0.0;
	float sum0001 = 0.0;
	float sum0010 = 0.0;
	float sum0011 = 0.0;
	float sum0100 = 0.0;
	float sum0101 = 0.0;
	float sum0110 = 0.0;
	float sum0111 = 0.0;
	float sum1000 = 0.0;
	float sum1001 = 0.0;
	float sum1010 = 0.0;
	float sum1011 = 0.0;
	float sum1100 = 0.0;
	float sum1101 = 0.0;
	float sum1110 = 0.0;
	float sum1111 = 0.0;
	unsigned short int current = 0;

	if (i<Ni && tx<Nk)		as00[0][tx][ty] = A[tx*Ni + i];
	else					as00[0][tx][ty] = 0.0;
	if (i+TILE_SIZE<Ni && tx<Nk)	as01[0][tx][ty] = A[tx*Ni + i+TILE_SIZE];
	else							as01[0][tx][ty] = 0.0;
	if (i+2*TILE_SIZE<Ni && tx<Nk)	as10[0][tx][ty] = A[tx*Ni + i+2*TILE_SIZE];
	else							as10[0][tx][ty] = 0.0;
	if (i+3*TILE_SIZE<Ni && tx<Nk)	as11[0][tx][ty] = A[tx*Ni + i+3*TILE_SIZE];
	else							as11[0][tx][ty] = 0.0;
	if (j<Nj && ty<Nk)		bs00[0][ty][tx] = B[ty*Nj + j];
	else					bs00[0][ty][tx] = 0.0;
	if (j+TILE_SIZE<Nj && ty<Nk)	bs01[0][ty][tx] = B[ty*Nj + j+TILE_SIZE];
	else							bs01[0][ty][tx] = 0.0;
	if (j+2*TILE_SIZE<Nj && ty<Nk)	bs10[0][ty][tx] = B[ty*Nj + j+2*TILE_SIZE];
	else							bs10[0][ty][tx] = 0.0;
	if (j+3*TILE_SIZE<Nj && ty<Nk)	bs11[0][ty][tx] = B[ty*Nj + j+3*TILE_SIZE];
	else							bs11[0][ty][tx] = 0.0;
	__syncthreads();

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0000 += as00[current][k-kt][ty]*bs00[current][k-kt][tx];
		}
		if (i<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0001 += as00[current][k-kt][ty]*bs01[current][k-kt][tx];
		}
		if (i<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0010 += as00[current][k-kt][ty]*bs10[current][k-kt][tx];
		}
		if (i<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0011 += as00[current][k-kt][ty]*bs11[current][k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0100 += as01[current][k-kt][ty]*bs00[current][k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0101 += as01[current][k-kt][ty]*bs01[current][k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0110 += as01[current][k-kt][ty]*bs10[current][k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0111 += as01[current][k-kt][ty]*bs11[current][k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1000 += as10[current][k-kt][ty]*bs00[current][k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1001 += as10[current][k-kt][ty]*bs01[current][k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1010 += as10[current][k-kt][ty]*bs10[current][k-kt][tx];
		}
		if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1011 += as10[current][k-kt][ty]*bs11[current][k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1100 += as11[current][k-kt][ty]*bs00[current][k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1101 += as11[current][k-kt][ty]*bs01[current][k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1110 += as11[current][k-kt][ty]*bs10[current][k-kt][tx];
		}
		if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1111 += as11[current][k-kt][ty]*bs11[current][k-kt][tx];
		}
		current = (current+1)&1;
		if (i<Ni && (kt+TILE_SIZE+tx)<Nk)		as00[current][tx][ty] = A[(kt+TILE_SIZE+tx)*Ni + i];
		else									as00[current][tx][ty] = 0.0;
		if (i+TILE_SIZE<Ni && (kt+TILE_SIZE+tx)<Nk)		as01[current][tx][ty] = A[(kt+TILE_SIZE+tx)*Ni + i+TILE_SIZE];
		else											as01[current][tx][ty] = 0.0;
		if (i+2*TILE_SIZE<Ni && (kt+TILE_SIZE+tx)<Nk)	as10[current][tx][ty] = A[(kt+TILE_SIZE+tx)*Ni + i+2*TILE_SIZE];
		else											as10[current][tx][ty] = 0.0;
		if (i+3*TILE_SIZE<Ni && (kt+TILE_SIZE+tx)<Nk)	as11[current][tx][ty] = A[(kt+TILE_SIZE+tx)*Ni + i+3*TILE_SIZE];
		else											as11[current][tx][ty] = 0.0;

		if (j<Nj && (kt+TILE_SIZE+ty)<Nk)		bs00[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j];
		else									bs00[current][ty][tx] = 0.0;
		if (j+TILE_SIZE<Nj && (kt+TILE_SIZE+ty)<Nk)		bs01[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j+TILE_SIZE];
		else											bs01[current][ty][tx] = 0.0;
		if (j+2*TILE_SIZE<Nj && (kt+TILE_SIZE+ty)<Nk)	bs10[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j+2*TILE_SIZE];
		else											bs10[current][ty][tx] = 0.0;
		if (j+3*TILE_SIZE<Nj && (kt+TILE_SIZE+ty)<Nk)	bs11[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j+3*TILE_SIZE];
		else											bs11[current][ty][tx] = 0.0;
		__syncthreads();
	}
	if (i<Ni && j<Nj)				C[i*Nj + j] = sum0000;
	if (i<Ni && j+TILE_SIZE<Nj)		C[i*Nj + j+TILE_SIZE] = sum0001;
	if (i<Ni && j+2*TILE_SIZE<Nj)	C[i*Nj + j+2*TILE_SIZE] = sum0010;
	if (i<Ni && j+3*TILE_SIZE<Nj)	C[i*Nj + j+3*TILE_SIZE] = sum0011;
	if (i+TILE_SIZE<Ni && j<Nj)				C[(i+TILE_SIZE)*Nj + j] = sum0100;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum0101;
	if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum0110;
	if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum0111;
	if (i+2*TILE_SIZE<Ni && j<Nj)				C[(i+2*TILE_SIZE)*Nj + j] = sum1000;
	if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)		C[(i+2*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1001;
	if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1010;
	if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1011;
	if (i+3*TILE_SIZE<Ni && j<Nj)				C[(i+3*TILE_SIZE)*Nj + j] = sum1100;
	if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)		C[(i+3*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1101;
	if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1110;
	if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1111;
}

__global__ void aTb_gpu_i2j2db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int i = blockIdx.y*blockDim.y*2 + ty;
	const unsigned int j = blockIdx.x*blockDim.x*2 + tx;

	__shared__ float as0[2][TILE_SIZE][TILE_SIZE];
	__shared__ float as1[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs0[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs1[2][TILE_SIZE][TILE_SIZE];
	float sum00 = 0.0;
	float sum01 = 0.0;
	float sum10 = 0.0;
	float sum11 = 0.0;
	unsigned short int current = 0;

	if (i<Ni && tx<Nk)	as0[0][tx][ty] = A[tx*Ni + i];
	else				as0[0][tx][ty] = 0.0;
	if (i+TILE_SIZE<Ni && tx<Nk)	as1[0][tx][ty] = A[tx*Ni + i+TILE_SIZE];
	else							as1[0][tx][ty] = 0.0;
	if (j<Nj && ty<Nk)	bs0[0][ty][tx] = B[ty*Nj + j];
	else				bs0[0][ty][tx] = 0.0;
	if (j+TILE_SIZE<Nj && ty<Nk)	bs1[0][ty][tx] = B[ty*Nj + j+TILE_SIZE];
	else							bs1[0][ty][tx] = 0.0;
	__syncthreads();
	
	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum00 += as0[current][k-kt][ty]*bs0[current][k-kt][tx];
		}
		if (i<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum01 += as0[current][k-kt][ty]*bs1[current][k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum10 += as1[current][k-kt][ty]*bs0[current][k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum11 += as1[current][k-kt][ty]*bs1[current][k-kt][tx];
		}
		current = (current+1)&1;
		if (i<Ni && (kt+TILE_SIZE+tx)<Nk)
			as0[current][tx][ty] = A[(kt+TILE_SIZE+tx)*Ni + i];
		else
			as0[current][tx][ty] = 0.0;
		if (i+TILE_SIZE<Ni && (kt+TILE_SIZE+tx)<Nk)
			as1[current][tx][ty] = A[(kt+TILE_SIZE+tx)*Ni + i+TILE_SIZE];
		else
			as1[current][tx][ty] = 0.0;
		if (j<Nj && (kt+TILE_SIZE+ty)<Nk)
			bs0[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j];
		else
			bs0[current][ty][tx] = 0.0;
		if (j+TILE_SIZE<Nj && (kt+TILE_SIZE+ty)<Nk)
			bs1[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j+TILE_SIZE];
		else
			bs1[current][ty][tx] = 0.0;
		__syncthreads();
	}
	if (i<Ni && j<Nj)	C[i*Nj + j] = sum00;
	if (i<Ni && j+TILE_SIZE<Nj)	C[i*Nj + j+TILE_SIZE] = sum01;
	if (i+TILE_SIZE<Ni && j<Nj)	C[(i+TILE_SIZE)*Nj + j] = sum10;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum11;
}

__global__ void aTb_gpu_i2j2(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int i = blockIdx.y*blockDim.y*2 + ty;
	const unsigned int j = blockIdx.x*blockDim.x*2 + tx;
	__shared__ float as0[TILE_SIZE][TILE_SIZE];
	__shared__ float as1[TILE_SIZE][TILE_SIZE];
	__shared__ float bs0[TILE_SIZE][TILE_SIZE];
	__shared__ float bs1[TILE_SIZE][TILE_SIZE];
	float sum00 = 0.0;
	float sum01 = 0.0;
	float sum10 = 0.0;
	float sum11 = 0.0;
	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && (kt+tx)<Nk)		as0[tx][ty] = A[(kt+tx)*Ni + i];
		else						as0[tx][ty] = 0.0;
		if (i+TILE_SIZE<Ni && (kt+tx)<Nk)	as1[tx][ty] = A[(kt+tx)*Ni + i+TILE_SIZE];
		else								as1[tx][ty] = 0.0;
		if (j<Nj && (kt+ty)<Nk)		bs0[ty][tx] = B[(kt+ty)*Nj + j];
		else						bs0[ty][tx] = 0.0;
		if (j+TILE_SIZE<Nj && (kt+ty)<Nk)	bs1[ty][tx] = B[(kt+ty)*Nj + j+TILE_SIZE];
		else								bs1[ty][tx] = 0.0;
		__syncthreads();

		if (i<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum00 += as0[k-kt][ty]*bs0[k-kt][tx];
		}
		if (i<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum01 += as0[k-kt][ty]*bs1[k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum10 += as1[k-kt][ty]*bs0[k-kt][tx];
		}
		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum11 += as1[k-kt][ty]*bs1[k-kt][tx];
		}
		__syncthreads();
	}
	if (i<Ni && j<Nj)	C[i*Nj + j] = sum00;
	if (i<Ni && j+TILE_SIZE<Nj)	C[i*Nj + j+TILE_SIZE] = sum01;
	if (i+TILE_SIZE<Ni && j<Nj)	C[(i+TILE_SIZE)*Nj + j] = sum10;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum11;
}

__global__ void aTb_gpu_db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int i = blockIdx.y*blockDim.y + ty;
	const unsigned int j = blockIdx.x*blockDim.x + tx;

	__shared__ float as[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs[2][TILE_SIZE][TILE_SIZE];
	float sum = 0;
	unsigned short int current = 0;

	if (i<Ni && tx<Nk)	as[0][tx][ty] = A[tx*Ni + i];
	else				as[0][tx][ty] = 0.0;
	if (j<Nj && ty<Nk)	bs[0][ty][tx] = B[ty*Nj + j];
	else				bs[0][ty][tx] = 0.0;
	__syncthreads();

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && j<Nj) {
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum += as[current][k-kt][ty]*bs[current][k-kt][tx];
		}
		current = (current+1)&1;
		if (i<Ni && (kt+TILE_SIZE+tx)<Nk)	as[current][tx][ty] = A[(kt+TILE_SIZE+tx)*Ni + i];
		else								as[current][tx][ty] = 0.0;
		if (j<Nj && (kt+TILE_SIZE+ty)<Nk)	bs[current][ty][tx] = B[(kt+TILE_SIZE+ty)*Nj + j];
		else								bs[current][ty][tx] = 0.0;
		__syncthreads();
	}
	if (i<Ni && j<Nj)	C[i*Nj + j] = sum;
}

__global__ void abT_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++) 
   for (j = 0; j < Nj; j++)
    C[i*Nj+j]=0.0;
  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[i][k]*B[j][k];
     C[i*Nj+j]=C[i*Nj+j]+A[i*Nk+k]*B[j*Nk+k];
}
// abT i4j4
__global__ void abT_gpu_i4j4(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned tx = threadIdx.x;
	const unsigned ty = threadIdx.y;
	const unsigned j = blockIdx.x*blockDim.x*4 + tx;
	const unsigned i = blockIdx.y*blockDim.y*4 + ty;
	float sum0000 = 0.0;
	float sum0001 = 0.0;
	float sum0010 = 0.0;
	float sum0011 = 0.0;
	float sum0100 = 0.0;
	float sum0101 = 0.0;
	float sum0110 = 0.0;
	float sum0111 = 0.0;
	float sum1000 = 0.0;
	float sum1001 = 0.0;
	float sum1010 = 0.0;
	float sum1011 = 0.0;
	float sum1100 = 0.0;
	float sum1101 = 0.0;
	float sum1110 = 0.0;
	float sum1111 = 0.0;

	__shared__ float as00[TILE_SIZE][TILE_SIZE];
	__shared__ float as01[TILE_SIZE][TILE_SIZE];
	__shared__ float as10[TILE_SIZE][TILE_SIZE];
	__shared__ float as11[TILE_SIZE][TILE_SIZE];
	__shared__ float bs00[TILE_SIZE][TILE_SIZE];
	__shared__ float bs01[TILE_SIZE][TILE_SIZE];
	__shared__ float bs10[TILE_SIZE][TILE_SIZE];
	__shared__ float bs11[TILE_SIZE][TILE_SIZE];

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && (kt+tx)<Nk)		as00[ty][tx] = A[i*Nk + kt+tx];
		else						as00[ty][tx] = 0.0;
		if (i+TILE_SIZE<Ni && (kt+tx)<Nk)	as01[ty][tx] = A[(i+TILE_SIZE)*Nk + kt+tx];
		else								as01[ty][tx] = 0.0;
		if (i+2*TILE_SIZE<Ni && (kt+tx)<Nk)	as10[ty][tx] = A[(i+2*TILE_SIZE)*Nk + kt+tx];
		else								as10[ty][tx] = 0.0;
		if (i+3*TILE_SIZE<Ni && (kt+tx)<Nk)	as11[ty][tx] = A[(i+3*TILE_SIZE)*Nk + kt+tx];
		else	as11[ty][tx] = 0.0;

		if (j<Nj && (kt+ty)<Nk)		bs00[tx][ty] = B[j*Nk + kt+ty];
		else						bs00[tx][ty] = 0.0;
		if (j+TILE_SIZE<Nj && (kt+ty)<Nk)	bs01[tx][ty] = B[(j+TILE_SIZE)*Nk + kt+ty];
		else								bs01[tx][ty] = 0.0;
		if (j+2*TILE_SIZE<Nj && (kt+ty)<Nk)	bs10[tx][ty] = B[(j+2*TILE_SIZE)*Nk + kt+ty];
		else								bs10[tx][ty] = 0.0;
		if (j+3*TILE_SIZE<Nj && (kt+ty)<Nk)	bs11[tx][ty] = B[(j+3*TILE_SIZE)*Nk + kt+ty];
		else								bs11[tx][ty] = 0.0;
		__syncthreads();

		if (i<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0000 += as00[ty][k-kt]*bs00[tx][k-kt];
		if (i<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0001 += as00[ty][k-kt]*bs01[tx][k-kt];
		if (i<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0010 += as00[ty][k-kt]*bs10[tx][k-kt];
		if (i<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0011 += as00[ty][k-kt]*bs11[tx][k-kt];
		if (i+TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0100 += as01[ty][k-kt]*bs00[tx][k-kt];
		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0101 += as01[ty][k-kt]*bs01[tx][k-kt];
		if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0110 += as01[ty][k-kt]*bs10[tx][k-kt];
		if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0111 += as01[ty][k-kt]*bs11[tx][k-kt];
		if (i+2*TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1000 += as10[ty][k-kt]*bs00[tx][k-kt];
		if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1001 += as10[ty][k-kt]*bs01[tx][k-kt];
		if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1010 += as10[ty][k-kt]*bs10[tx][k-kt];
		if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1011 += as10[ty][k-kt]*bs11[tx][k-kt];
		if (i+3*TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1100 += as11[ty][k-kt]*bs00[tx][k-kt];
		if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1101 += as11[ty][k-kt]*bs01[tx][k-kt];
		if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1110 += as11[ty][k-kt]*bs10[tx][k-kt];
		if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1111 += as11[ty][k-kt]*bs11[tx][k-kt];
		__syncthreads();
	}
	if (i<Ni && j<Nj)				C[i*Nj + j] = sum0000;
	if (i<Ni && j+TILE_SIZE<Nj)		C[i*Nj + j+TILE_SIZE] = sum0001;
	if (i<Ni && j+2*TILE_SIZE<Nj)	C[i*Nj + j+2*TILE_SIZE] = sum0010;
	if (i<Ni && j+3*TILE_SIZE<Nj)	C[i*Nj + j+3*TILE_SIZE] = sum0011;
	if (i+TILE_SIZE<Ni && j<Nj)				C[(i+TILE_SIZE)*Nj + j] = sum0100;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum0101;
	if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum0110;
	if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum0111;
	if (i+2*TILE_SIZE<Ni && j<Nj)				C[(i+2*TILE_SIZE)*Nj + j] = sum1000;
	if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)		C[(i+2*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1001;
	if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1010;
	if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1011;
	if (i+3*TILE_SIZE<Ni && j<Nj)				C[(i+3*TILE_SIZE)*Nj + j] = sum1100;
	if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)		C[(i+3*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1101;
	if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1110;
	if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1111;
}

__global__ void abT_gpu_i2j2db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned tx = threadIdx.x;
	const unsigned ty = threadIdx.y;
	const unsigned j = blockIdx.x*blockDim.x*2 + tx;
	const unsigned i = blockIdx.y*blockDim.y*2 + ty;
	float sum00 = 0.0;
	float sum01 = 0.0;
	float sum10 = 0.0;
	float sum11 = 0.0;
	unsigned short int current = 0;
	__shared__ float as0[2][TILE_SIZE][TILE_SIZE];
	__shared__ float as1[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs0[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs1[2][TILE_SIZE][TILE_SIZE];
	if (i<Ni && tx<Nk)	as0[0][ty][tx] = A[i*Nk + tx];
	else				as0[0][ty][tx] = 0.0;
	if (i+TILE_SIZE<Ni && tx<Nk)	as1[0][ty][tx] = A[(i+TILE_SIZE)*Nk + tx];
	else							as1[0][ty][tx] = 0.0;
	if (j<Nj && ty<Nk)	bs0[0][tx][ty] = B[j*Nk + ty];
	else				bs0[0][tx][ty] = 0.0;
	if (j+TILE_SIZE<Nj && ty<Nk)	bs1[0][tx][ty] = B[(j+TILE_SIZE)*Nk + ty];
	else							bs1[0][tx][ty] = 0.0;
	__syncthreads();

	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum00 += as0[current][ty][k-kt]*bs0[current][tx][k-kt];
		if (i<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum01 += as0[current][ty][k-kt]*bs1[current][tx][k-kt];
		if (i+TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum10 += as1[current][ty][k-kt]*bs0[current][tx][k-kt];
		if (i<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum11 += as1[current][ty][k-kt]*bs1[current][tx][k-kt];

		current = (current+1)&1;
		if (i<Ni && (kt+TILE_SIZE+tx)<Nk)	as0[current][ty][tx] = A[i*Nk + kt+TILE_SIZE+tx];
		else								as0[current][ty][tx] = 0.0;
		if (i+TILE_SIZE<Ni && (kt+TILE_SIZE+tx)<Nk)	as1[current][ty][tx] = A[(i+TILE_SIZE)*Nk + kt+TILE_SIZE+tx];
		else										as1[current][ty][tx] = 0.0;
		if (j<Nj && (kt+TILE_SIZE+ty)<Nk)	bs0[current][tx][ty] = B[j*Nk + kt+TILE_SIZE+ty];
		else								bs0[current][tx][ty] = 0.0;
		if (j+TILE_SIZE<Nj && (kt+TILE_SIZE+ty)<Nk)	bs1[current][tx][ty] = B[(j+TILE_SIZE)*Nk + kt+TILE_SIZE+ty];
		else										bs1[current][tx][ty] = 0.0;
		__syncthreads();
	}
	if (i<Ni && j<Nj)	C[i*Nj + j] = sum00;
	if (i<Ni && j+TILE_SIZE<Nj)	C[i*Nj + j+TILE_SIZE] = sum01;
	if (i+TILE_SIZE<Ni && j<Nj)	C[(i+TILE_SIZE)*Nj + j] = sum10;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum11;
}

__global__ void abT_gpu_db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned tx = threadIdx.x;
	const unsigned ty = threadIdx.y;
	const unsigned j = blockIdx.x*blockDim.x + tx;
	const unsigned i = blockIdx.y*blockDim.y + ty;
	float sum = 0;
	unsigned short int current = 0;
	__shared__ float as[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs[2][TILE_SIZE][TILE_SIZE];
	if (i<Ni && tx<Nk)	as[0][ty][tx] = A[i*Nk + tx];
	else				as[0][ty][tx] = 0.0;
	if (j<Nj && ty<Nk)	bs[0][tx][ty] = B[j*Nk + ty];
	else				bs[0][tx][ty] = 0.0;
	__syncthreads();
	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum += as[current][ty][k-kt]*bs[current][tx][k-kt];

		current = (current+1)&1;
		if (i<Ni && (kt+TILE_SIZE+tx)<Nk)	as[current][ty][tx] = A[i*Nk + kt+TILE_SIZE+tx];
		else								as[current][ty][tx] = 0.0;
		if (j<Nj && (kt+TILE_SIZE+ty)<Nk)	bs[current][tx][ty] = B[j*Nk + kt+TILE_SIZE+ty];
		else								bs[current][tx][ty] = 0.0;
		__syncthreads();
	}
	if (i<Ni && j<Nj)	C[i*Nj + j] = sum;
}

__global__ void aTbT_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++) 
   for (j = 0; j < Nj; j++)
    C[i*Nj+j]=0.0;
  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[k][i]*B[j][k];
     C[i*Nj+j]=C[i*Nj+j]+A[k*Ni+i]*B[j*Nk+k];
}

__global__ void aTbT_gpu_i4j4(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int j = blockIdx.x*blockDim.x*4 + tx;
	const unsigned int i = blockIdx.y*blockDim.y*4 + ty;
	float sum0000 = 0.0;
	float sum0001 = 0.0;
	float sum0010 = 0.0;
	float sum0011 = 0.0;
	float sum0100 = 0.0;
	float sum0101 = 0.0;
	float sum0110 = 0.0;
	float sum0111 = 0.0;
	float sum1000 = 0.0;
	float sum1001 = 0.0;
	float sum1010 = 0.0;
	float sum1011 = 0.0;
	float sum1100 = 0.0;
	float sum1101 = 0.0;
	float sum1110 = 0.0;
	float sum1111 = 0.0;
	__shared__ float as00[TILE_SIZE][TILE_SIZE];
	__shared__ float as01[TILE_SIZE][TILE_SIZE];
	__shared__ float as10[TILE_SIZE][TILE_SIZE];
	__shared__ float as11[TILE_SIZE][TILE_SIZE];
	__shared__ float bs00[TILE_SIZE][TILE_SIZE];
	__shared__ float bs01[TILE_SIZE][TILE_SIZE];
	__shared__ float bs10[TILE_SIZE][TILE_SIZE];
	__shared__ float bs11[TILE_SIZE][TILE_SIZE];
	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && (kt+tx)<Nk)				as00[tx][ty] = A[(kt+tx)*Ni + i];
		else								as00[tx][ty] = 0.0;
		if (i+TILE_SIZE<Ni && (kt+tx)<Nk)	as01[tx][ty] = A[(kt+tx)*Ni + i+TILE_SIZE];
		else								as01[tx][ty] = 0.0;
		if (i+2*TILE_SIZE<Ni && (kt+tx)<Nk)	as10[tx][ty] = A[(kt+tx)*Ni + i+2*TILE_SIZE];
		else								as10[tx][ty] = 0.0;
		if (i+3*TILE_SIZE<Ni && (kt+tx)<Nk)	as11[tx][ty] = A[(kt+tx)*Ni + i+3*TILE_SIZE];
		else								as11[tx][ty] = 0.0;
		if (j<Nj && (kt+ty)<Nk)				bs00[tx][ty] = B[j*Nk + kt+ty];
		else								bs00[tx][ty] = 0.0;
		if (j+TILE_SIZE<Nj && (kt+ty)<Nk)	bs01[tx][ty] = B[(j+TILE_SIZE)*Nk + kt+ty];
		else								bs01[tx][ty] = 0.0;
		if (j+2*TILE_SIZE<Nj && (kt+ty)<Nk)	bs10[tx][ty] = B[(j+2*TILE_SIZE)*Nk + kt+ty];
		else								bs10[tx][ty] = 0.0;
		if (j+3*TILE_SIZE<Nj && (kt+ty)<Nk)	bs11[tx][ty] = B[(j+3*TILE_SIZE)*Nk + kt+ty];
		else								bs11[tx][ty] = 0.0;
		__syncthreads();
		if (i<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0000 += as00[k-kt][ty]*bs00[tx][k-kt];
		if (i<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0001 += as00[k-kt][ty]*bs01[tx][k-kt];
		if (i<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0010 += as00[k-kt][ty]*bs10[tx][k-kt];
		if (i<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0011 += as00[k-kt][ty]*bs11[tx][k-kt];
		if (i+TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0100 += as01[k-kt][ty]*bs00[tx][k-kt];
		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0101 += as01[k-kt][ty]*bs01[tx][k-kt];
		if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0110 += as01[k-kt][ty]*bs10[tx][k-kt];
		if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum0111 += as01[k-kt][ty]*bs11[tx][k-kt];
		if (i+2*TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1000 += as10[k-kt][ty]*bs00[tx][k-kt];
		if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1001 += as10[k-kt][ty]*bs01[tx][k-kt];
		if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1010 += as10[k-kt][ty]*bs10[tx][k-kt];
		if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1011 += as10[k-kt][ty]*bs11[tx][k-kt];
		if (i+3*TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1100 += as11[k-kt][ty]*bs00[tx][k-kt];
		if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1101 += as11[k-kt][ty]*bs01[tx][k-kt];
		if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1110 += as11[k-kt][ty]*bs10[tx][k-kt];
		if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum1111 += as11[k-kt][ty]*bs11[tx][k-kt];
		__syncthreads();
	}
	if (i<Ni && j<Nj)				C[i*Nj + j] = sum0000;
	if (i<Ni && j+TILE_SIZE<Nj)		C[i*Nj + j+TILE_SIZE] = sum0001;
	if (i<Ni && j+2*TILE_SIZE<Nj)	C[i*Nj + j+2*TILE_SIZE] = sum0010;
	if (i<Ni && j+3*TILE_SIZE<Nj)	C[i*Nj + j+3*TILE_SIZE] = sum0011;
	if (i+TILE_SIZE<Ni && j<Nj)				C[(i+TILE_SIZE)*Nj + j] = sum0100;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum0101;
	if (i+TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum0110;
	if (i+TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum0111;
	if (i+2*TILE_SIZE<Ni && j<Nj)				C[(i+2*TILE_SIZE)*Nj + j] = sum1000;
	if (i+2*TILE_SIZE<Ni && j+TILE_SIZE<Nj)		C[(i+2*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1001;
	if (i+2*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1010;
	if (i+2*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+2*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1011;
	if (i+3*TILE_SIZE<Ni && j<Nj)				C[(i+3*TILE_SIZE)*Nj + j] = sum1100;
	if (i+3*TILE_SIZE<Ni && j+TILE_SIZE<Nj)		C[(i+3*TILE_SIZE)*Nj + j+TILE_SIZE] = sum1101;
	if (i+3*TILE_SIZE<Ni && j+2*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+2*TILE_SIZE] = sum1110;
	if (i+3*TILE_SIZE<Ni && j+3*TILE_SIZE<Nj)	C[(i+3*TILE_SIZE)*Nj + j+3*TILE_SIZE] = sum1111;
}

__global__ void aTbT_gpu_i2j2(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int j = blockIdx.x*blockDim.x*2 + tx;
	const unsigned int i = blockIdx.y*blockDim.y*2 + ty;
	float sum00 = 0.0;
	float sum01 = 0.0;
	float sum10 = 0.0;
	float sum11 = 0.0;
	__shared__ float as0[TILE_SIZE][TILE_SIZE];
	__shared__ float as1[TILE_SIZE][TILE_SIZE];
	__shared__ float bs0[TILE_SIZE][TILE_SIZE];
	__shared__ float bs1[TILE_SIZE][TILE_SIZE];
	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && (kt+tx)<Nk)				as0[tx][ty] = A[(kt+tx)*Ni + i];
		else								as0[tx][ty] = 0.0;
		if (i+TILE_SIZE<Ni && (kt+tx)<Nk)	as1[tx][ty] = A[(kt+tx)*Ni + i+TILE_SIZE];
		else								as1[tx][ty] = 0.0;
		if (j<Nj && (kt+ty)<Nk)				bs0[tx][ty] = B[j*Nk + kt+ty];
		else								bs0[tx][ty] = 0.0;
		if (j+TILE_SIZE<Nj && (kt+ty)<Nk)	bs1[tx][ty] = B[(j+TILE_SIZE)*Nk + kt+ty];
		else								bs1[tx][ty] = 0.0;
		__syncthreads();
		if (i<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum00 += as0[k-kt][ty]*bs0[tx][k-kt];
		if (i<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum01 += as0[k-kt][ty]*bs1[tx][k-kt];
		if (i+TILE_SIZE<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum10 += as1[k-kt][ty]*bs0[tx][k-kt];
		if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum11 += as1[k-kt][ty]*bs1[tx][k-kt];
		__syncthreads();
	}
	if (i<Ni && j<Nj)			C[i*Nj + j] = sum00;
	if (i<Ni && j+TILE_SIZE<Nj)	C[i*Nj + j+TILE_SIZE] = sum01;
	if (i+TILE_SIZE<Ni && j<Nj)	C[(i+TILE_SIZE)*Nj + j] = sum10;
	if (i+TILE_SIZE<Ni && j+TILE_SIZE<Nj)	C[(i+TILE_SIZE)*Nj + j+TILE_SIZE] = sum11;
}

__global__ void aTbT_gpu_db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int j = blockIdx.x*blockDim.x + tx;
	const unsigned int i = blockIdx.y*blockDim.y + ty;
	float sum = 0;
	unsigned short int current = 0;
	__shared__ float as[2][TILE_SIZE][TILE_SIZE];
	__shared__ float bs[2][TILE_SIZE][TILE_SIZE];
	if (i<Ni && tx<Nk)	as[0][tx][ty] = A[tx*Ni + i];
	else				as[0][tx][ty] = 0.0;
	if (j<Nj && ty<Nk)	bs[0][tx][ty] = B[j*Nk + ty];
	else				bs[0][tx][ty] = 0.0;
	__syncthreads();
	for (int kt=0; kt<Nk; kt+=TILE_SIZE) {
		if (i<Ni && j<Nj)
			for (int k=kt; k<MIN(kt+TILE_SIZE,Nk); k++)
				sum += as[current][k-kt][ty]*bs[current][tx][k-kt];

		current = (current+1)&1;
		if (i<Ni && (kt+TILE_SIZE+tx)<Nk)	as[current][tx][ty] = A[(kt+TILE_SIZE+tx)*Ni + i];
		else								as[current][tx][ty] = 0.0;
		if (j<Nj && (kt+TILE_SIZE+ty)<Nk)	bs[current][tx][ty] = B[j*Nk + kt+TILE_SIZE+ty];
		else								bs[current][tx][ty] = 0.0;
		__syncthreads();
	}
	if (i<Ni && j<Nj)	C[i*Nj + j] = sum;
}

