
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#define threshold 0.0001
#define BLOCK_SIZE 16
#define FIXME 1

void checkCUDAError(const char *msg);

hipEvent_t start, stop;
float tstart, elapsedTime;

__global__ void ab_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void ab_gpu_i4j4db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void ab_gpu_i4j4(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void ab_gpu_i2j2db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void ab_gpu_db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);

__global__ void aTb_gpu_i4j4(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void aTb_gpu_i4j4db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void aTb_gpu_i2j2db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void aTb_gpu_i2j2(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void aTb_gpu_db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void aTb_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);

__global__ void abT_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void abT_gpu_i4j4(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void abT_gpu_i2j2db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void abT_gpu_db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);

__global__ void aTbT_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void aTbT_gpu_i4j4(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void aTbT_gpu_i2j2(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void aTbT_gpu_db(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);

void ab_seq(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[i][k]*B[k][j];
     C[i*Nj+j]=C[i*Nj+j]+A[i*Nk+k]*B[k*Nj+j];
}

void abT_seq(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[i][k]*B[j][k];
     C[i*Nj+j]=C[i*Nj+j]+A[i*Nk+k]*B[j*Nk+k];
}

void aTb_seq(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[k][i]*B[k][j];
     C[i*Nj+j]=C[i*Nj+j]+A[k*Ni+i]*B[k*Nj+j];
}

void aTbT_seq(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[k][i]*B[j][k];
     C[i*Nj+j]=C[i*Nj+j]+A[k*Ni+i]*B[j*Nk+k];
}


int main(){

  float *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k;
  int Ni,Nj,Nk;


  printf("Specify Matrix dimension Ni, Nj, Nk: ");
  scanf("%d %d %d", &Ni,&Nj,&Nk);
  printf("\nNi=%d\tNj=%d\tNk=%d\n", Ni, Nj, Nk);

  h_A = (float *) malloc(sizeof(float)*Ni*Nk);
  h_B = (float *) malloc(sizeof(float)*Nk*Nj);
  h_C = (float *) malloc(sizeof(float)*Ni*Nj);
  h_Cref = (float *) malloc(sizeof(float)*Ni*Nj);;

  for (i=0; i<Ni; i++)
   for (k=0; k<Nk; k++)
    h_A[k*Ni+i] = rand();
  for (k=0; k<Nk; k++)
   for (j=0; j<Nj; j++)
    h_B[k*Nj+j] = rand();

  
 // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, Ni*Nk*sizeof(float));
  hipMalloc(&d_B, Nk*Nj*sizeof(float));
  hipMalloc(&d_C, Ni*Nj*sizeof(float));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, Ni*Nk*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Nk*Nj*sizeof(float), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D transfer failure");

  dim3 block(BLOCK_SIZE, BLOCK_SIZE);  
  dim3 grid((Nj+BLOCK_SIZE-1)/BLOCK_SIZE, (Ni+BLOCK_SIZE-1)/BLOCK_SIZE);
  dim3 grid2((Nj+2*BLOCK_SIZE-1)/(2*BLOCK_SIZE), (Ni+2*BLOCK_SIZE-1)/(2*BLOCK_SIZE));
  dim3 grid4((Nj+4*BLOCK_SIZE-1)/(4*BLOCK_SIZE), (Ni+4*BLOCK_SIZE-1)/(4*BLOCK_SIZE));


  //int version = 3;
  //int version = 2;
  //int version = 1;
  //int version = 0;
  for(int version=0; version<4; version++)
  {
   for(i=0;i<Ni;i++) for(j=0;j<Nj;j++) h_Cref[i*Nj+j] = 0;
   switch (version) {
      case 0: ab_seq(h_A,h_B,h_Cref,Ni,Nj,Nk);  break;
      case 1: aTb_seq(h_A,h_B,h_Cref,Ni,Nj,Nk); break;
      case 2: abT_seq(h_A,h_B,h_Cref,Ni,Nj,Nk); break;
      case 3: aTbT_seq(h_A,h_B,h_Cref,Ni,Nj,Nk);
    }
	float gflops;
	float max_gflops = -0.1;
    for(int trial=0;trial<3;trial++)
    {
     for(i=0;i<Ni;i++) for(j=0;j<Nj;j++) h_C[i*Nj+j] = 0; 
      printf("Trial %d: ",trial);
	
		int minn = ((Ni<=Nj)? Ni:Nj);

      hipEventCreate(&start);
      hipEventCreate(&stop);
      //cudaEventRecord(start);
      // Launch kernel
      switch (version) {
      case 0: 
		if (minn>=Nk) {
			hipEventRecord(start);
			ab_gpu_i4j4<<<grid4, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}else if (64==(Nk/minn)) {
			hipEventRecord(start);
			ab_gpu_i2j2db<<<grid2, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}else {
			hipEventRecord(start);
			ab_gpu_db<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}
		//ab_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
		printf("AB ");
		break;
      case 1:
		if (minn>=Nk) {
			hipEventRecord(start);
			aTb_gpu_i4j4<<<grid4, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}else if (64==(Nk/minn)) {
			hipEventRecord(start);
			aTb_gpu_i2j2db<<<grid2, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}else if (27==(Nk/minn)) {
			hipEventRecord(start);
			aTb_gpu_i2j2<<<grid2, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}else {
			hipEventRecord(start);
			aTb_gpu_db<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}
		//aTb_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
		printf("ATB ");
		break;
      case 2:
		if (minn>=Nk) {
			hipEventRecord(start);
			abT_gpu_i4j4<<<grid4, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}else if (Nk/minn==64) {
			hipEventRecord(start);
			abT_gpu_i2j2db<<<grid2, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}else {
			hipEventRecord(start);
			abT_gpu_db<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}
		//abT_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
		printf("ABT ");
		break;
      case 3:
		if (minn > Nk) {
			hipEventRecord(start);
			aTbT_gpu_i4j4<<<grid4, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}else if ((Ni==Nk) || (Nk/Ni==64)) {
			hipEventRecord(start);
			aTbT_gpu_i2j2<<<grid2, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}else {
			hipEventRecord(start);
			aTbT_gpu_db<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk);
			checkCUDAError("GPU kernel launch failure");
			hipEventRecord(stop);
			hipEventSynchronize(stop);
		}
		//aTbT_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); 
		printf("ATBT ");
		break;
      }
      //checkCUDAError("GPU kernel launch failure");
      //cudaEventRecord(stop);
      //cudaEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start,stop);
      hipDeviceSynchronize();
      // Copy results back to host
      hipMemcpy(h_C, d_C, Ni*Nj*sizeof(float), hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpy D2H");
      for (int i = 0; i < Ni*Nj; i++) if (fabs((h_C[i]-h_Cref[i])/h_Cref[i])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", i, h_C[i], h_Cref[i]); return -1;}
      gflops = 2.0e-6*Ni*Nj*Nk/elapsedTime;
	  //printf("GFLOPS: \t%.2f\n",2.0e-6*Ni*Nj*Nk/elapsedTime);
	  printf("GFLOPS: \t%.2f\n", gflops);
	  if (gflops > max_gflops)	max_gflops = gflops;
     }
	 printf("MAX GFLOPS: \t\t%.2f\n", max_gflops);
	 printf("\n");
  }
  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

